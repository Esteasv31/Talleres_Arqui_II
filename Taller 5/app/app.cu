#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

// size of the square matrix
#define N 4

// GPU kernel
__global__ 
void saxpy(int x, int *A, int *B, int *C){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
        int j = threadIdx.y + blockDim.y * blockIdx.y;
        while (j < ((N*3)+1)){
                C[i+j] = x * A[i+j] + B[i+j];
                j++;
        }

}

//CPU function
void _saxpy(int x, int *A1, int *B1, int *C1){
      	for (int i = 0; i < N; ++i){
		C1[i] = x * A1[i] + B1[i];
	}
}

int main(int argc, char **argv){
	
	int a[N][N], b[N][N], c[N][N], c2[N][N];
 	int *dev_a, *dev_b, *dev_c;
 	int cont,i,j;

  	/* inicializando variables con datos foo*/
 	for (i = 0; i < N; i++) {
  		cont = 0;
  		for (j = 0; j < N; j++) {
   			a[i][j] = cont;
   			b[i][j] = cont;
   			cont++;
  		}
 	}

  	int size = N * N * sizeof(int);

  	hipMalloc((void **) &dev_a, size);
 	hipMalloc((void **) &dev_b, size);
 	hipMalloc((void **) &dev_c, size);

  	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
 	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

  	dim3 dimGrid(1, 1);
 	dim3 dimBlock(N, N);

	clock_t start_d=clock();
    	printf("Doing GPU Vector app\n\n");
  	saxpy<<<dimGrid, dimBlock>>>(N * 3, dev_a, dev_b, dev_c);

	//Wait for kernel call to finish
    	hipDeviceSynchronize();

	clock_t end_d = clock();
    
    	printf("Doing CPU Vector app\n\n");
    	clock_t start_h = clock();
    	_saxpy(N * 3, (int *)a, (int *)b, (int *)c);
    	clock_t end_h = clock();

	//Time computing
    	double time_d = (double)(end_d-start_d)/CLOCKS_PER_SEC;
    	double time_h = (double)(end_h-start_h)/CLOCKS_PER_SEC;

  	hipMemcpy(c2, dev_c, size, hipMemcpyDeviceToHost);
	printf("n = %d \t GPU time = %fs \t CPU time = %fs\n\n", N, time_d, time_h);

  	hipFree(dev_a);
 	hipFree(dev_b);
 	hipFree(dev_c);

	printf("-----------------------------------------------------------------------------------------------------------------------\n");

  	// imprimiendo
 	for (int y = 0; y < N; y++) {
  		for (int x = 0; x < N; x++) {
   			printf("[%d][%d]=%d ", y, x, c[y][x]);
  		}
  		printf("\n");
 	}

	printf("-----------------------------------------------------------------------------------------------------------------------\n");

	// imprimiendo
 	for (int y = 0; y < N; y++) {
  		for (int x = 0; x < N; x++) {
   			printf("[%d][%d]=%d ", y, x, c2[y][x]);
  		}
  		printf("\n");
 	}

  	return 0;
}

	


