#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

// size of the square matrix
#define N 4

//Cuda error checking - non mandatory
void cudaCheckError() {
	hipError_t e=hipGetLastError();
	if(e!=hipSuccess) {
		printf("Cuda failure %s:%d:'%s'\n",__FILE__,__LINE__,hipGetErrorString(e));
  		exit(0);
 	}
}

// GPU kernel
__global__ void matrixMul(int *A, int *B, int *C){
	int k;
	int sum = 0;
	int col = threadIdx.x + blockDim.x * blockIdx.x;	
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	if (col < N && row < N) {
  		for (k = 0; k < N; k++) {
   			sum += A[row * N + k] * B[k * N + col];
  		}
  		C[row * N + col] = sum;
 	}
}

//CPU function
void matrix_Mul(int A1[][N], int B1[][N], int C1[][N]){
        int n = 0;
	int m = 0;
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
   			int sum = 0;
			for (int k = 0; k < N; k++) {
    				m = A1[i][k];
    				n = B1[k][j];
    				sum += m * n;
   			}
   			C1[i][j] = sum;
  		}
 	}
}

int main(int argc, char **argv){
	
	int a[N][N], b[N][N], c[N][N], c2[N][N];
 	int *dev_a, *dev_b, *dev_c;
 	int cont,i,j;

  	/* inicializando variables con datos foo*/
 	for (i = 0; i < N; i++) {
  		cont = 0;
  		for (j = 0; j < N; j++) {
   			a[i][j] = cont;
   			b[i][j] = cont;
   			cont++;
  		}
 	}

  	int size = N * N * sizeof(int);

  	hipMalloc((void **) &dev_a, size);
 	hipMalloc((void **) &dev_b, size);
 	hipMalloc((void **) &dev_c, size);

  	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
 	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

  	dim3 dimGrid(1, 1);
 	dim3 dimBlock(N, N);

	clock_t start_d=clock();
    	printf("Doing GPU Vector mulMatrix\n\n");
  	matrixMul<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c);
	cudaCheckError();

	//Wait for kernel call to finish
    	hipDeviceSynchronize();

	clock_t end_d = clock();
    
    	printf("Doing CPU Vector mulMatrix\n\n");
    	clock_t start_h = clock();
    	matrix_Mul(a, b, c);
    	clock_t end_h = clock();

	//Time computing
    	double time_d = (double)(end_d-start_d)/CLOCKS_PER_SEC;
    	double time_h = (double)(end_h-start_h)/CLOCKS_PER_SEC;

  	hipMemcpy(c2, dev_c, size, hipMemcpyDeviceToHost);
	printf("n = %d \t GPU time = %fs \t CPU time = %fs\n\n", N, time_d, time_h);

  	hipFree(dev_a);
 	hipFree(dev_b);
 	hipFree(dev_c);

	printf("-----------------------------------------------------------------------------------------------------------------------\n");

  	// imprimiendo
 	for (int y = 0; y < N; y++) {
  		for (int x = 0; x < N; x++) {
   			printf("[%d][%d]=%d ", y, x, c[y][x]);
  		}
  		printf("\n");
 	}

	printf("-----------------------------------------------------------------------------------------------------------------------\n");

	// imprimiendo
 	for (int y = 0; y < N; y++) {
  		for (int x = 0; x < N; x++) {
   			printf("[%d][%d]=%d ", y, x, c2[y][x]);
  		}
  		printf("\n");
 	}

  	return 0;
}

	


